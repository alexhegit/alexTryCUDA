/*
 * Author: Ye HE
 * Data: 4/18/2020
 * 
 * Description: Query information of the GPU
 * 
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <stdint.h>
 #include <string.h>
 #include <iostream>
 #include <ctype.h>

 #include "hip/hip_runtime.h"
 
 

#ifndef ul
#define ul unsigned long
#endif

#ifndef ui
#define ui unsigned int
#endif

void chkCUDAErr(hipError_t ErrorID)
{
    if (ErrorID != hipSuccess) {
        printf("CUDA ERROR :::%s\n", hipGetErrorString(ErrorID));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv)
{
    hipError_t cudaStatus;
    hipDeviceProp_t GPUprop;
    ul SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;
    char SupportedBlocks[100];

    ui GPUid = 0;
    char ProgName[255]; 

    int NumGPUs = 0;
    hipGetDeviceCount(&NumGPUs);
    if (NumGPUs == 0) {
        printf("\n No CUDA Device is available\n");
        exit(EXIT_FAILURE);
    }

    strcpy(ProgName, "QueryGPU");
    switch (argc) {
        case 2: 
            GPUid = atoi(argv[1]);
            break;
        default:
            printf("\n\nUsage: %s [GPU ID]", ProgName);
            printf("\n\nExample: %s 0", ProgName);
            printf("\n\n");
            exit(EXIT_FAILURE);
    }

    printf("Query the info of the GPU #%d\n", GPUid);    
    cudaStatus = hipSetDevice(GPUid);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! no ==No CUDA-capable GPU installed?");
        exit(EXIT_FAILURE);
    }

    hipGetDeviceProperties(&GPUprop, GPUid);
	SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
	SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%lu %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks, (SupportedMBlocks >= 5) ? 'M' : 'K');
    MaxThrPerBlk = (ul)GPUprop.maxThreadsPerBlock;

    printf("GPU Name: %s\n", GPUprop.name);
    printf("SupportedBlocks: %s, MaxThrPerBlk: %lu\n", SupportedBlocks, MaxThrPerBlk);
    for (int i = 0; i < 3; i++) {
        printf("maxGridSize[%d]=%d  ", i, GPUprop.maxGridSize[i]);
    }
    printf("\n");
    printf("SharedMemPerBlock:%ld Bytes, SharedMemPerMultiprocessor: %ld Bytes\n", GPUprop.sharedMemPerBlock, GPUprop.sharedMemPerMultiprocessor);
    printf("Total Global Memory: %ld MiB\n", GPUprop.totalGlobalMem/1024/1024);
    printf("Unified Addressing: %d\n", GPUprop.unifiedAddressing);

    printf("WarpSize:%d\n", GPUprop.warpSize);

    return 0;
}

