
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloFromCPU()
{
    printf("Hello World from CPU!\n");
}

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(void)
{
    helloFromCPU();
    helloFromGPU <<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}