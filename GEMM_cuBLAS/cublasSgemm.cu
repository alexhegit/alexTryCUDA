/*
cuda中用cublas库做矩阵乘法
这里矩阵C=A*B，原始文档给的公式是C=alpha*A*B+beta*C，所以这里alpha=1，beta=0。

主要使用cublasSgemm这个函数，这个函数的第二个参数有三种类型，这里CUBLAS_OP_N求出来矩阵结果是按行排列，所以不需要转置了。

如果用CUBLAS_OP_T参数求得的结果是按列排列，做成C形式的矩阵应该还需要转置一下，并且后面跟的参数也不太一样，这个参数我就没再尝试了。

代码如下：
*/

#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>

using namespace std;

int main()
{
    srand(time(0));
    int M = 2;            //矩阵A的行，矩阵C的行
    int N = 3;            //矩阵A的列，矩阵B的行
    int K = 4;            //矩阵B的列，矩阵C的列

    float *h_A = (float*)malloc(sizeof(float)*M*N);
    float *h_B = (float*)malloc(sizeof(float)*N*K);
    float *h_C = (float*)malloc(sizeof(float)*M*K);

    for (int i = 0; i < M*N; i++)
    {
        h_A[i] = rand() % 10;
        cout << h_A[i] << "  ";
        if ((i + 1) % N == 0)
            cout << endl;        
    }
    cout << endl;

    for (int i = 0; i < N*K; i++)
    {
        h_B[i] = rand() % 10;
        cout << h_B[i] << "  ";
        if ((i + 1) % K == 0)
            cout << endl;
    }
    cout << endl;

    float *d_A, *d_B, *d_C,*d_CT;
    hipMalloc((void**)&d_A, sizeof(float)*M*N);
    hipMalloc((void**)&d_B, sizeof(float)*N*K);
    hipMalloc((void**)&d_C, sizeof(float)*M*K);

    hipMemcpy(d_A, h_A, M*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N*K * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1;
    float beta = 0;

    //C=A*B
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,   
        K,                    //矩阵B的列数
        M,                    //矩阵A的行数
        N,                    //矩阵A的列数
        &alpha,           
        d_B,            
        K,                    
        d_A,         
        N,         
        &beta,          
        d_C,           
        K);

    hipMemcpy(h_C, d_C, M*K * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < M*K; i++)
    {
        cout << h_C[i] << "  ";
        if ((i+1)%K==0)
            cout << endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}